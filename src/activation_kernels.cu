#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "activations.h"
#include "hip/hip_runtime.h"
}

/*
 * light tanh activate function: 
 *   y = 0.001 * x              if x < 0;
 *   y = x                      if 0 <= x <= 1;
 *   y = 0.001 * (x - 1) + 1    else.
 *
 * the grident of light tanh:
 *   y' = 1                     if 0 <= x <= 1;
 *   y' = 0.001                 else.
*/
__device__ float lhtan_activate_kernel(float x) 
{
    if(x < 0) return 0.001f * x;
    if(x > 1) return 0.001f * (x - 1) + 1;
    return x; 
}

__device__ float lhtan_gradient_kernel(float x)
{
    if(x >= 0 && x <= 1) return 1;
    return 0.001f;
}

/*
 * hard tan activate function:
 *   y = -1                     if x < -1;
 *   y = 1                      if x > 1;
 *   y = x                      else.
 *
 * the grident of hard tan:
 *   y = 1                      if -1 <= x <= 1;
 *   y = 0                      else.
*/
__device__ float hardtan_activate_kernel(float x)
{
    if(x < -1) return -1;
    if(x > 1) return 1;
    return x;
    if(x >= -1 && x <= 1) return 1;
    return 0;
}

__device__ float hardtan_gradient_kernel(float x)
{
    if(x >= -1 && x <= 1) return 1;
    return 0;
}

/*
 * linear activate function:
 *   y = x
 *
 * the grident of linear:
 *   y' = 1
*/
__device__ float linear_activate_kernel(float x) {return x;}
__device__ float linear_gradient_kernel(float x) {return 1;}

__device__ float logistic_activate_kernel(float x) {return 1.f/(1.f + expf(-x));}
__device__ float logistic_gradient_kernel(float x) {return expf(-x) / ((1 + expf(-x)) * (1 + expf(-x)));}

__device__ float relu_activate_kernel(float x) {return x*(x > 0);}
__device__ float relu_gradient_kernel(float x) {return x > 0;}

__device__ float elu_activate_kernel(float x) {return (x >= 0)*x + (x < 0)*(expf(x)-1);}
__device__ float elu_gradient_kernel(float x) {return (x >= 0) + (x < 0)*(x + 1);}

__device__ float selu_activate_kernel(float x) {return (x >= 0)*1.0507f*x + (x < 0)*1.0507f*1.6732*(expf(x)-1);}
__device__ float selu_gradient_kernel(float x) {return (x >= 0)*1.0507f + (x < 0)*(1.0507f*1.6732 + x);}


__device__ float leaky_activate_kernel(float x) {return (x > 0)*x + (x <= 0)*0.1f*x;}
__device__ float leaky_gradient_kernel(float x) {return (x > 0) + (x <= 0)*0.1f;}

__device__ float tanh_activate_kernel(float x) {return (exp(2*x)-1)/(exp(2*x)+1);}
__device__ float tanh_gradient_kernel(float x) {return 1 - x*x;}


// 根据传入的激活函数选项选择具体的激活函数
__device__ float activate_kernel(float x, ACTIVATION a)
{
    switch (a)
    {
    case LHTAN:         return lhtan_activate_kernel(x);
    case HARDTAN:       return hardtan_activate_kernel(x);
    case LINEAR:        return linear_activate_kernel(x);
    case LOGISTIC:      return logistic_activate_kernel(x);
    case RELU:          return relu_activate_kernel(x);
    case ELU:           return elu_activate_kernel(x);
    case SELU:          return selu_activate_kernel(x);
    case LEAKY:         return leaky_activate_kernel(x);
    case TANH:          return tanh_activate_kernel(x);
    }
}
// 同时计算其梯度
__device__ float gradient_kernel(float x, ACTIVATION a)
{
    switch (a)
    {
    case LHTAN:         return lhtan_gradient_kernel(x);
    case HARDTAN:       return hardtan_gradient_kernel(x);
    case LINEAR:        return linear_gradient_kernel(x);
    case LOGISTIC:      return logistic_gradient_kernel(x);
    case RELU:          return relu_gradient_kernel(x);
    case ELU:           return elu_gradient_kernel(x);
    case SELU:          return selu_gradient_kernel(x);
    case LEAKY:         return leaky_gradient_kernel(x);
    case TANH:          return tanh_gradient_kernel(x);
    }
}

/*
* x: 某一层网络的输入
* dy: x 对应的微分
* dx: x对应的输入的微分
* s: 二维矩阵x的第一个维度
* 
*/
__global__ void binary_gradient_array_kernel(float *x, float *dy, int n, int s, float *dx)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    int i = id % s; //获取当前的
    int b = id / s;
    float x1 = x[b*s + i];
    float x2 = x[b*s + s/2 + i];
    if(id < n) {
        float de = dy[id];
        dx[b*s + i] = x2 * de;
        dx[b*s + s/2 + i] = x1 * de;
    }
}

extern "C" void binary_gradient_array_gpu(float *x, float *dx, int n, int s, float *y)
{
    binary_gradient_array_kernel<<<cuda_gridsize(n/2), BLOCK>>>(x, dx, n, s, y);
    check_error(hipPeekAtLastError());
}


__global__ void binary_activate_array_kernel(float *x, int n, int s, float *y)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    int i = id % s;
    int b = id / s;
    float x1 = x[b*s + i];
    float x2 = x[b*s + s/2 + i];
    if(id < n) y[id] = x1 * x2;
}

extern "C" void binary_activate_array_gpu(float *x, float *dx, int n, int s, float *y)
{
    binary_activate_array_kernel<<<cuda_gridsize(n/2), BLOCK>>>(x, dx, n, s, y);
    check_error(hipPeekAtLastError());
}
